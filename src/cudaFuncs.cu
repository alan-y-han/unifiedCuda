#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>
#include <cstring>
#include <utility>
#include "cudaUtils.h"

#include <eigen3/Eigen/Dense>
#include <eigen3/Eigen/src/Core/Matrix.h>
#include <hipblas.h>

#ifdef __JETBRAINS_IDE__
#define threadIdx();
#define blockIdx();
#define blockDim();
#endif

#if (defined __GNUC__) && (__GNUC__>4 || __GNUC_MINOR__>=7)
#undef _GLIBCXX_ATOMIC_BUILTINS
#undef _GLIBCXX_USE_INT128
#endif

//class Managed
//{
//public:
//    void* operator new(size_t len)
//    {
//        void* ptr;
//        hipMallocManaged(&ptr, len);
//        hipDeviceSynchronize();
//        return ptr;
//    }
//
//    void operator delete(void* ptr)
//    {
//        hipDeviceSynchronize();
//        hipFree(ptr);
//    }
//};
//
//class String : public Managed
//{
//    int length;
//    char *data;
//
//public:
//    String (const String& s)
//    {
//        length = s.length;
//        hipMallocManaged(&data, length);
//        memcpy(data, s.data, length);
//    }
//};
//
//class dataElem : public Managed
//{
//public:
//    int prop1;
//    int prop2;
//    String name;
//};

//__global__ void cube(float * d_out, float * d_in){
//    int id = threadIdx.x;
//    float value = d_in[id];
//    d_out[id] = value * value * value;
//}

//void printCubes()
//{
//    std::cout << "Hello, World!" << std::endl;
//    const int ARRAY_SIZE = 96;
//    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);
//
//    // generate the input array on the host
//    float* h_in;
//    checkCudaErrors(hipMallocManaged(&h_in, ARRAY_BYTES));
//    for (int i = 0; i < ARRAY_SIZE; i++) {
//        h_in[i] = i;
//    }
//    float* h_out;
//    checkCudaErrors(hipMallocManaged(&h_out, ARRAY_BYTES));
//
//    // launch the kernel
//    cube<<<1, ARRAY_SIZE>>>(h_out, h_in);
//    checkCudaErrors(hipGetLastError());
//    checkCudaErrors(hipDeviceSynchronize());
//
//    // print out the resulting array
//    for (int i = 0; i < ARRAY_SIZE; i++) {
//        printf("%f", h_out[i]);
//        printf(((i % 4) != 3) ? "\t" : "\n");
//    }
//
//    checkCudaErrors(hipFree(h_in));
//    checkCudaErrors(hipFree(h_out));
//}

//__global__ void cudaDoEigen(Eigen::MatrixXd* m, int rows, int columns)
//{
//    printf("CUDA value: %lf\n", (*m)(0,0));
//}

__global__ void cudaDoEigen(double* m, int rows, int columns)
{
//    Eigen::Matrix stuff = m;
//    printf("CUDA testing\n");
    printf("CUDA ptr: %p\n", m);
    printf("CUDA value: %lf\n", m[0]);
    printf("CUDA value: %lf\n", m[1]);
    printf("CUDA value: %lf\n", m[2]);
    printf("CUDA value: %lf\n", m[3]);

}

void doEigenStuff()
{
    double* m_pointer;
    checkCudaErrors(hipMallocManaged(&m_pointer, sizeof(double) * 4));

//    m_pointer[0] = 1;

//    for (int i = 0; i < 4; i++)
//    {
//
//    }
//
    for (int j = 0; j < 4; j++)
    {
        m_pointer[j] = static_cast<double>(j);
    }
//    new (m_pointer) Eigen::MatrixXd(2,2);

    Eigen::MatrixXd m = Eigen::Map<Eigen::MatrixXd>(m_pointer, 2, 2);

//    Eigen::MatrixXd& m = *((Eigen::MatrixXd*)m_pointer);
    m(0,0) = 3;
    m(1,0) = 2.5;
    m(0,1) = -1;
    m(1,1) = m(1,0) + m(0,1);
//
    std::cout << m << std::endl;
//
    for (int i = 0; i < 4; i++)
    {
        std::cout << "CPU value: " << m_pointer[i] << std::endl;
    }



//    cudaDoEigen<<<1, 1>>>(m_pointer, 2, 2);
//    hipDeviceSynchronize();

//    checkCudaErrors(hipGetLastError());
}
