#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "cudaUtils.h"

__global__ void cube(float * d_out, float * d_in){
    int id = threadIdx.x;
    float value = d_in[id];
    d_out[id] = value * value * value;
}

void printCubes()
{
    std::cout << "Hello, World!" << std::endl;
    const int ARRAY_SIZE = 96;
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

    // generate the input array on the host
    float* h_in;
    checkCudaErrors(hipMallocManaged(&h_in, ARRAY_BYTES));
    for (int i = 0; i < ARRAY_SIZE; i++) {
        h_in[i] = i;
    }
    float* h_out;
    checkCudaErrors(hipMallocManaged(&h_out, ARRAY_BYTES));

    // launch the kernel
    cube<<<1, ARRAY_SIZE>>>(h_out, h_in);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // print out the resulting array
    for (int i =0; i < ARRAY_SIZE; i++) {
        printf("%f", h_out[i]);
        printf(((i % 4) != 3) ? "\t" : "\n");
    }

    checkCudaErrors(hipFree(h_in));
    checkCudaErrors(hipFree(h_out));
}